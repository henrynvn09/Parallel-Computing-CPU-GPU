#include "hip/hip_runtime.h"
// Header inclusions, if any...
#include "lib/cnn.cuh"
#include "cnn_gpu.cuh"

// Using declarations, if any...

__global__ void cnn_gpu(
    float* input,
    float* weight,
    float* bias,
    float* output)
{
  // Your code goes here...
}
